
#include <hip/hip_runtime.h>
/**
Reduction using warp reduction instructions. This approach uses less
shared memory than previous approaches.
 */
 __inline__ __device__ float warpReduce(float value) {
    // Use XOR mode to perform butterfly reduction
    for (int i=16; i>=1; i/=2)
        value += __shfl_xor_sync(0xffffffff, value, i, 32);

    // "value" now contains the sum across all threads
    //printf("Thread %d final value = %d\n", threadIdx.x, value);
    return value;
}

__inline__ __device__ float blockReduce(float sum) {
    sum = warpReduce(sum);
    int tid = threadIdx.x;
    __shared__ float psums[16];
    if (tid % 32 == 0) {
        psums[tid / 32] = sum;
    }
    __syncthreads();

    sum = 0;
    for (int i = 0; i < blockDim.x / 32; i++) {
        sum += psums[i];
    }
    return sum;
}

extern "C" __global__
void kernel(int M, int N, int C, float* X, float* centers, int* labels) {
    int m = blockIdx.x;
    int n = threadIdx.x;

    float x = X[m*N + n];
    float min_dis = 1000000;
    float cluster = -1;
    for (int i = 0; i < C; i++) {
        float c = centers[i*N + n];
        float pdis = (x-c) * (x-c);
        float dis = blockReduce(pdis);
        if (dis < min_dis) {
            min_dis = dis;
            cluster = i;
        }
    }

    if (n == 0) {
        labels[m] = cluster;
    }
}