
#include <hip/hip_runtime.h>
/**
Reduction using warp reduction instructions. This approach uses less
shared memory than previous approaches.
 */
 __inline__ __device__ float warpReduce(float value) {
    // Use XOR mode to perform butterfly reduction
    for (int i=16; i>=1; i/=2)
        value += __shfl_xor_sync(0xffffffff, value, i, 32);

    // "value" now contains the sum across all threads
    //printf("Thread %d final value = %d\n", threadIdx.x, value);
    return value;
}

__inline__ __device__ float blockReduce(float sum) {
    sum = warpReduce(sum);
    int tid = threadIdx.x;
    __shared__ float psums[16];
    if (tid % 32 == 0) {
        psums[tid / 32] = sum;
    }
    __syncthreads();

    sum = 0;
    for (int i = 0; i < blockDim.x / 32; i++) {
        sum += psums[i];
    }
    return sum;
}

extern "C" __global__
void kernel(int M, int N, int C, int BLOCK_M, float* X, float* centers) {
    int m = blockIdx.x;
    int n = threadIdx.x;
    
    if (m > M) {
        return;
    }

    float x = X[m*N + n];
    float min_dist = 100000000;
    int min_dist_cluster = -1;
    for (int i = 0; i < C; i++) {
        float c = centers[i*N + n];
        float t = (x-c) * (x-c);

        float sum = blockReduce(t);
        if (n == 0) {
            float distance = sqrt(sum);
            if (distance < min_dist) {
                min_dist = distance;
                min_dist_cluster = i;
            }
        }
    }
    
}